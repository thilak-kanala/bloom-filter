#include "hip/hip_runtime.h"
#include "utility.cuh"

int main(int argc, char const *argv[])
{
    hipFree(0);

    print_gpu_info();

    N_WORDS_TO_INSERT = atoi(argv[1]);

    /* Input */
    INPUT_DATA input_data;
    input_data.words = (char *)malloc((N_WORDS_TO_INSERT + 1) * MAX_WORD_BYTES * sizeof(char));
    input_data.words[0] = ' ';
    input_data.words[1] = '\0';
    input_data.spaces = (int *)malloc((N_WORDS_TO_INSERT + 2) * sizeof(int));

    prepare_input_data(input_data, INPUT_WORDS_FILE);
    print_input_data(input_data);

    /* Bloom Filter */
    struct BLOOM_FILTER bf;
    prepare_bloom_filter(&bf, N_WORDS_TO_INSERT, FALSE_PROBABILITY);
    print_bloom_filter(&bf);

    /* GPU */
    INPUT_DATA d_input_data;
    BLOOM_FILTER d_bf;

    // Testing
    // int *h_arr = (int*) malloc(10 * sizeof(int));
    // int *d_arr;

    // hipMalloc((void**)&d_arr, 10 * sizeof(int));
    // hipMemcpy(d_arr, h_arr, 10*sizeof(int), hipMemcpyHostToDevice);

    // kernel<<<1, 32>>>(d_arr);

    // hipFree(d_arr);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("1.CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMalloc((void **)&d_input_data.words, sizeof(input_data.words));
    hipMalloc((void **)&d_input_data.spaces, sizeof(input_data.spaces));
    hipMalloc((void **)&d_bf.bf, sizeof(bf.bf));
    hipMalloc((void **)&d_bf.m, sizeof(bf.m));
    hipMalloc((void **)&d_bf.n, sizeof(bf.n));
    hipMalloc((void **)&d_bf.p, sizeof(bf.p));
    hipMalloc((void **)&d_bf.k, sizeof(bf.k));

    // check for error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("2.CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemcpy(d_input_data.words, input_data.words, sizeof(input_data.words), hipMemcpyHostToDevice);
    hipMemcpy(d_input_data.spaces, input_data.spaces, sizeof(input_data.spaces), hipMemcpyHostToDevice);
    hipMemcpy(d_bf.bf, bf.bf, sizeof(bf.bf), hipMemcpyHostToDevice);
    hipMemcpy(&d_bf.m, &bf.m, sizeof(bf.m), hipMemcpyHostToDevice);
    hipMemcpy(&d_bf.n, &bf.n, sizeof(bf.n), hipMemcpyHostToDevice);
    hipMemcpy(&d_bf.p, &bf.p, sizeof(bf.p), hipMemcpyHostToDevice);
    hipMemcpy(&d_bf.k, &bf.k, sizeof(bf.k), hipMemcpyHostToDevice);


    // check for error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("3.CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    // TODO: change dimensions
    insert_words_kernel<<<ceil(bf.n / 256.0), 256>>>(d_input_data, d_bf);

    // check for error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("4.CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Copy the bloom filter from GPU to CPU
    hipMemcpy(bf.bf, d_bf.bf, sizeof(d_bf.bf), hipMemcpyDeviceToHost);

    write_bloom_filter(&bf, BLOOM_FILTER_FILE);

    // check for error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("5.CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }    

    hipFree(d_input_data.words);
    hipFree(d_input_data.spaces);
    hipFree(d_bf.bf);
    hipFree(d_bf.m);
    hipFree(d_bf.n);
    hipFree(d_bf.p);
    hipFree(d_bf.k);

    // check for error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("6.CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return 0;
}