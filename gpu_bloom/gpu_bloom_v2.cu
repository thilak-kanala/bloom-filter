#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <stddef.h>   /* uint32_t, NULL */
#include <stdint.h>   /* uint8_t, uint32_t, uint64_t */
#include <inttypes.h> /* print formatting */

#include "xxhash64.cuh"

#define BLOOM_FILTER_SIZE 335477044
#define N_HASHES 23
#define FILE_WORDS_TO_INSERT "words_insert.txt"
#define FILE_WORDS_TO_QUERY "words_query.txt"

uint32_t total_words = 0;

__device__ uint32_t djb2(char *words_to_insert, int len_words_to_insert);
__device__ uint32_t jenkins(char *words_to_insert, int len_words_to_insert);

__device__ __host__ uint64_t XXH64(void const *const input, uint32_t const length, uint64_t const seed);

__device__ __host__ void print_bloom_filter(uint32_t *bloom_filter);

/* === BIT VECTOR BEGIN ===*/

// sets the kth bit in bitvector 'bv'
__device__
void set_bit(uint32_t *bv, uint32_t k)
{
    bv[k / 32] |= (1 << (k % 32));
}

// clears the kth bit in bitvector 'bv'
__device__ __host__
void clear_bit(uint32_t *bv, uint32_t k)
{
    bv[k / 32] &= ~(1 << (k % 32));
}

__device__ __host__
int test_bit(uint32_t *bv, uint32_t k)
{
    return ((bv[k / 32] & (1 << (k % 32))) != 0);
}

/* === BIT VECTOR END ===*/



__device__ __host__ void print_binary_64(uint64_t n, const char *message)
{
    printf("%s", message);
    int binary[64];
    int bi = 63;
    for (int i = 0; i < 64; i++)
    {
        uint64_t mask = 1;
        if (n & (mask << i))
        {
            binary[bi--] = 1;
        }
        else
        {
            binary[bi--] = 0;
        }
    }

    for (int i = 0; i < 64; i++)
    {
        if (i % 4 == 0)
        {
            printf("\t");
        }
        printf("%d", binary[i]);
    }
    printf("\n");
}

__device__ __host__ void print_binary_32(uint64_t n, const char *message)
{
    printf("%s", message);
    int binary[32];
    int bi = 31;
    for (int i = 0; i < 32; i++)
    {
        uint64_t mask = 1;
        if (n & (mask << i))
        {
            binary[bi--] = 1;
        }
        else
        {
            binary[bi--] = 0;
        }
    }

    for (int i = 0; i < 32; i++)
    {
        if (i % 4 == 0)
        {
            printf("\t");
        }
        printf("%d", binary[i]);
    }
    printf("\n");
}

/* Splits the 64bit hash into 2 32 bit hashes h1 and h2 */
__device__ __host__ void split_hash_bits(uint64_t hash, uint64_t *h1, uint64_t *h2)
{
    uint64_t mask;
    uint64_t one_64bit = 1;

    // Clear all bits
    mask = 0;

    // Create mask to extract bottom 32 bits
    for (int i = 0; i < 32; i++)
    {
        mask |= (one_64bit << i);
    }

    // Extract bottom 32 bits
    *h1 = hash & mask;

    // Create mask to extract top 32 bits
    for (int i = 32; i < 64; i++)
    {
        mask |= (one_64bit << i);
    }

    // Extract top 32 bits
    *h2 = (hash & mask) >> 32;
}

/* Splits the 32bit hash into 2 16 bit hashes h1 and h2 */
__device__ __host__ void split_hash_bits_32(uint64_t hash, uint32_t *h1, uint32_t *h2)
{
    uint32_t mask;
    uint32_t one_32bit = 1;

    // Clear all bits
    mask = 0;

    // Create mask to extract bottom 16 bits
    for (int i = 0; i < 16; i++)
    {
        mask |= (one_32bit << i);
    }

    // Extract bottom 16 bits
    *h1 = hash & mask;

    // Create mask to extract top 16 bits
    for (int i = 16; i < 32; i++)
    {
        mask |= (one_32bit << i);
    }

    // Extract top 16 bits
    *h2 = (hash & mask) >> 16;
}


/* Generate a hash based on The Combinatorial Approach (https://github.com/Claudenw/BloomFilter/wiki/Bloom-Filters----An-Overview) */
__device__ void bloom_insert(uint32_t *d_bloom_filter, char *word, uint32_t word_len, int tid)
{
    uint32_t index = 0;

    uint64_t hash = XXH64((char*) word, word_len, 0);
    uint64_t h1;
    uint64_t h2;

    split_hash_bits(hash, &h1, &h2);
    // split_hash_bits_32(hash, &h1, &h2);

    for (uint32_t i = 0; i < N_HASHES; i++)
    {
        h1 += (h2 * i);
        h1 = h1 % BLOOM_FILTER_SIZE;
        index = (uint32_t)h1;

        // set index bit
        atomicOr(&d_bloom_filter[index / 32], (1 << (index % 32)));
    }
}

/* Generate a hash based on The Combinatorial Approach (https://github.com/Claudenw/BloomFilter/wiki/Bloom-Filters----An-Overview) */
__device__ void bloom_query(uint32_t *d_bloom_filter, char *word, uint32_t word_len, uint32_t *d_query_results, int tid)
{
    int is_present = 1;
    uint32_t index;
    uint32_t bloom_filter_partial;

    uint64_t hash = XXH64((char*) word, word_len, 0);
    uint64_t h1;
    uint64_t h2;

    split_hash_bits(hash, &h1, &h2);

    for (uint32_t i = 0; i < N_HASHES; i++)
    {
        h1 += (h2 * i);
        h1 = h1 % BLOOM_FILTER_SIZE;
        index = (uint32_t)h1;

        // extract the relevant part (32 bits) of bloom filter
        bloom_filter_partial = d_bloom_filter[index / 32];

        if ((bloom_filter_partial & (1 << (index % 32))) == 0)
        {
            is_present = 0;
            break;
        }
    }

    if (is_present)
    {
        // set the tid bit to indicate the word processed by tid is present
        atomicOr(&d_query_results[tid / 32], (1 << (tid % 32)));
    }
}

__global__ void map_bloom_kernel(char *d_words_to_insert, int len_words_to_insert, uint32_t *d_word_indices, uint32_t *d_bloom_filter, uint32_t total_words)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid >= total_words)
    {
        return;
    }

    // Find word start and end indices
    uint32_t word_len = d_word_indices[tid + 1] - d_word_indices[tid] - 1;
    uint32_t si = d_word_indices[tid] + 1;
    uint32_t ei = d_word_indices[tid + 1] - 1;

    // char *word = (char *)malloc((ei - si + 1) * sizeof(char));
    char word[128];

    for (uint32_t i = si; si <= ei; si++)
    {
        word[si - i] = d_words_to_insert[si];
    }

    // Add word to bloom filter
    bloom_insert(d_bloom_filter, word, word_len, tid);
    // free(word);
}

__global__ void query_bloom_kernel(char *d_words_to_query, int len_words_to_insert, uint32_t *d_word_indices, uint32_t *d_bloom_filter, uint32_t *d_query_results, uint32_t total_words)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // printf("%d: query kernel\n", tid);

    if (tid >= total_words)
    {
        return;
    }

    // Find word start and end indices
    uint32_t word_len = d_word_indices[tid + 1] - d_word_indices[tid] - 1;
    uint32_t si = d_word_indices[tid] + 1;
    uint32_t ei = d_word_indices[tid + 1] - 1;

    // char *word = (char *)malloc((ei - si + 1) * sizeof(char));
    char word[128];

    for (uint32_t i = si; si <= ei; si++)
    {
        // printf("tid, bid, si: %d, %d, %d\n", threadIdx.x, blockIdx.x, si);
        word[si - i] = d_words_to_query[si];
    }

    // Add word to bloom filter
    bloom_query(d_bloom_filter, word, word_len, d_query_results, tid);
    // free(word);
}

// Hash Functions
__device__
uint32_t
djb2(char *words_to_insert, int len_words_to_insert)
{
    uint32_t hash = 5381;
    char c;
    uint32_t i = 0;
    while (i++ < len_words_to_insert)
    {
        c = *words_to_insert++;
        hash = ((hash << 5) + hash) + c;
    }
    return hash;
}

__device__
uint32_t
jenkins(char *words_to_insert, int len_words_to_insert)
{
    uint32_t hash = 5381;
    uint32_t i = 0;
    while (i < len_words_to_insert)
    {
        hash += *words_to_insert++;
        hash += (hash << 10);
        hash ^= (hash >> 6);
        i++;
    }
    hash += (hash << 3);
    hash ^= (hash >> 11);
    hash += (hash << 15);
    return hash;
}

__device__ __host__ void print_bloom_filter(uint32_t *bloom_filter)
{
    // printf("Bloom Filter: ");
    // for (int i = 0; i < BLOOM_FILTER_SIZE; i++)
    // {
    //     if (test_bit(bloom_filter, i))
    //     {
    //         printf("1, ");
    //     }
    //     else
    //     {
    //         printf("0, ");
    //     }
    // }
    // printf("\n");

    // printf("The follwing bits are set in the bloom filter: ");
    // for (int i = 0; i < BLOOM_FILTER_SIZE; i++)
    // {
    //     if (test_bit(bloom_filter, i))
    //     {
    //         printf("%d, ", i);
    //     }
    // }
    // printf("\n\n");

    uint32_t count = 0;
    for (int i = 0; i < BLOOM_FILTER_SIZE; i++)
    {
        if (test_bit(bloom_filter, i))
        {
            count++;
        }
    }
    printf("%d bits are set in the bloom filter\n\n", count);
}

void print_query_results(const char *words_to_insert, int len_strin_in, uint32_t *word_indices, uint32_t *query_results, int total_words)
{
    printf("\n");

    int present = 0;

    for (int i = 0; i < total_words; i++)
    {
        // Find word start and end indices
        int word_len = word_indices[i + 1] - word_indices[i] - 1;
        int si = word_indices[i] + 1;
        int ei = word_indices[i + 1] - 1;

        char *word = (char *)malloc((ei - si + 2) * sizeof(char));
        for (int j = si; si <= ei; si++)
        {
            word[si - j] = words_to_insert[si];
        }
        word[word_len] = '\0';

        if (test_bit(query_results, i))
        {
            // printf("%s: present\n", word);
            present += 1;
        }
        else
        {
            // printf("%s: absent\n", word);
        }
    }

    printf("Query Result: %d / %d are present\n\n", present, total_words);
}

char *read_from_file(char *file_location)
{
    FILE *fp;
    uint32_t lSize;
    char *buffer;

    fp = fopen(file_location, "r");
    if( !fp ) 
    {
        perror(file_location);
        exit(1);
    }

    fseek(fp, 0L, SEEK_END);
    lSize = ftell(fp);
    rewind(fp);

    /* allocate memory for entire content */
    buffer = (char *)calloc(1, lSize + 1);
    if (!buffer)
        fclose(fp), fputs("memory alloc fails", stderr), exit(1);

    /* copy the file into the buffer */
    if (1 != fread(buffer, lSize, 1, fp))
        fclose(fp), free(buffer), fputs("entire read fails", stderr), exit(1);

    fclose(fp);

    return buffer;
}

int main(void)
{
    hipFree(0);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device name: %s\n", prop.name);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* === Intialize profiling related variables === */
    // hipEvent_t start, stop;
    // float time;

    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    /* === Map Words to Bloom Filter === */

    char* words_to_insert = read_from_file(FILE_WORDS_TO_INSERT);
    uint32_t len_words_to_insert = strlen(words_to_insert);

    uint32_t *word_indices;
    int wi = -1;
    total_words = 0;

    uint32_t *bloom_filter = (uint32_t *)calloc(ceil(BLOOM_FILTER_SIZE / 32.0), sizeof(uint32_t));

    for (uint32_t i = 0; i < len_words_to_insert; i++)
    {
        if (words_to_insert[i] == ' ')
        {
            total_words++;
        }
    }
    // To account for the space at the beginning and the end
    total_words -= 1;

    printf("\n---\n");
    printf("Number of items to insert: %d\n", total_words);
    printf("Number of bits in the Bloom Filter: %d\n", BLOOM_FILTER_SIZE);
    printf("Number of hash functions: %d\n", N_HASHES);
    printf("---\n");

    word_indices = (uint32_t *)calloc((total_words + 1), sizeof(uint32_t));

    for (uint32_t i = 0; i < len_words_to_insert; i++)
    {
        if (words_to_insert[i] == ' ')
        {
            word_indices[++wi] = i;
        }
    }

    printf("\n == Before Inserting Words ==\n");
    print_bloom_filter(bloom_filter);

    char *d_words_to_insert;
    uint32_t *d_word_indices;
    uint32_t *d_bloom_filter;

    hipEventRecord(start);
    hipMalloc((void **)&d_words_to_insert, len_words_to_insert * sizeof(char));
    hipMalloc((void **)&d_word_indices, (total_words + 1) * sizeof(uint32_t));
    hipMalloc((void **)&d_bloom_filter, ceil(BLOOM_FILTER_SIZE / 32.0) * sizeof(uint32_t));

    hipMemcpy(d_words_to_insert, words_to_insert, len_words_to_insert * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, (total_words + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_bloom_filter, bloom_filter, ceil(BLOOM_FILTER_SIZE / 32.0) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Transferring strings and bloom filter from Host to GPU Global Memory took %f ms\n", milliseconds);

    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("1.1.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    // hipEventRecord( start, 0 );

    hipEventRecord(start);
    map_bloom_kernel<<<ceil(total_words / 256.0), 256>>>(d_words_to_insert, len_words_to_insert, d_word_indices, d_bloom_filter, total_words);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Inserting words to Bloom Filter (kernel) took:  %f ms\n", milliseconds);

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("1.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(start);
    hipMemcpy(bloom_filter, d_bloom_filter, ceil(BLOOM_FILTER_SIZE / 32.0) * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Transferring Bloom Filter from GPU Global Memory to Host took %f ms\n", milliseconds);


    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("2.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    printf("\n == After Inserting Words to Bloom Filter ==\n");
    print_bloom_filter(bloom_filter);

    // printf("== Inserting words using gpu took %f ms ==\n", time);

    hipFree(d_words_to_insert);
    hipFree(d_word_indices);

    /* === Query Bloom Filter === */

    char *words_to_query = read_from_file(FILE_WORDS_TO_QUERY);
    uint32_t len_words_to_query = strlen(words_to_query);

    total_words = 0;
    for (uint32_t i = 0; i < len_words_to_query; i++)
    {
        if (words_to_query[i] == ' ')
        {
            total_words++;
        }
    }
    // To account for the space at the beginning and the end
    total_words -= 1;

    word_indices = (uint32_t *)calloc((total_words + 1), sizeof(int));

    wi = -1;
    for (uint32_t i = 0; i < len_words_to_query; i++)
    {
        if (words_to_query[i] == ' ')
        {
            word_indices[++wi] = i;
        }
    }

    uint32_t *query_results = (uint32_t *)calloc(total_words, sizeof(uint32_t));
    uint32_t *d_query_results;
    char *d_words_to_query;

    hipEventRecord(start);

    hipMalloc((void **)&d_words_to_query, len_words_to_query * sizeof(char));
    hipMalloc((void **)&d_word_indices, (total_words + 1) * sizeof(int));
    hipMalloc((void **)&d_query_results, (ceil(total_words / 32.0) * sizeof(uint32_t)));


    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("4.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipMemcpy(d_words_to_query, words_to_query, len_words_to_query * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, (total_words + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_query_results, query_results, (ceil(total_words / 32.0) * sizeof(uint32_t)), hipMemcpyHostToDevice);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Transerring Query Words to Global Memory took:  %f ms\n", milliseconds);

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("5.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(start);

    query_bloom_kernel<<<ceil(total_words / 256.0), 256>>>(d_words_to_query, len_words_to_query, d_word_indices, d_bloom_filter, d_query_results, total_words);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Querying words to Bloom Filter (kernel) took:  %f ms\n", milliseconds);

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("6.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(start);
    hipMemcpy(query_results, d_query_results, (ceil(total_words / 32.0) * sizeof(uint32_t)), hipMemcpyDeviceToHost);
        hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Transferring query results to host took:  %f ms\n", milliseconds);

    printf("\n== After Querying Words from Bloom Filter ==\n");

    print_query_results(words_to_query, len_words_to_query, word_indices, query_results, total_words);

    hipFree(d_words_to_query);
    hipFree(d_word_indices);
    hipFree(d_query_results);
    hipFree(d_bloom_filter);

    return 0;
}