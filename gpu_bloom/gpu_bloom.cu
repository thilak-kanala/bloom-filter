#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <stddef.h>   /* uint32_t, NULL */
#include <stdint.h>   /* uint8_t, uint32_t, uint64_t */
#include <inttypes.h> /* print formatting */

#include "xxhash64.cuh"
#include "hash.cuh"
#include "utility.cuh"
#include "defines.cuh"

uint32_t total_words = 0;

/*
    1 : XXHASH64
    2 : djb2
    3 : jenkins
*/
int HASH_FUNCTION = 1;

/* Generate a hash based on The Combinatorial Approach (https://github.com/Claudenw/BloomFilter/wiki/Bloom-Filters----An-Overview) */
__device__ void bloom_insert(uint32_t *d_bloom_filter, char *word, uint32_t word_len, int tid, int HASH_FUNCTION)
{
    uint32_t index = 0;

    // uint64_t hash = XXH64((char*) word, word_len, 0);
    // uint64_t h1;
    // uint64_t h2;

    // split_hash_bits(hash, &h1, &h2);
    // split_hash_bits_32(hash, &h1, &h2);

    uint32_t hash_value;
    hash_value = hash(word, word_len, HASH_FUNCTION);

    uint32_t h1;
    uint32_t h2;

    split_hash_bits_32(hash_value, &h1, &h2);

    for (uint32_t i = 0; i < N_HASHES; i++)
    {
        h1 += (h2 * i);
        h1 = h1 % BLOOM_FILTER_SIZE;
        index = h1;

        // set index bit
        atomicOr(&d_bloom_filter[index / 32], (1 << (index % 32)));
    }
}

/* Generate a hash based on The Combinatorial Approach (https://github.com/Claudenw/BloomFilter/wiki/Bloom-Filters----An-Overview) */
__device__ void bloom_query(uint32_t *d_bloom_filter, char *word, uint32_t word_len, uint32_t *d_query_results, int tid, int HASH_FUNCTION)
{
    int is_present = 1;
    uint32_t index;
    uint32_t bloom_filter_partial;

    // uint64_t hash = XXH64((char*) word, word_len, 0);
    // uint64_t h1;
    // uint64_t h2;

    // split_hash_bits(hash, &h1, &h2);

    uint32_t hash_value;
    hash_value = hash(word, word_len, HASH_FUNCTION);

    uint32_t h1;
    uint32_t h2;

    split_hash_bits_32(hash_value, &h1, &h2);

    for (uint32_t i = 0; i < N_HASHES; i++)
    {
        h1 += (h2 * i);
        h1 = h1 % BLOOM_FILTER_SIZE;
        index = h1;

        // extract the relevant part (32 bits) of bloom filter
        bloom_filter_partial = d_bloom_filter[index / 32];

        if ((bloom_filter_partial & (1 << (index % 32))) == 0)
        {
            is_present = 0;
            break;
        }
    }

    if (is_present)
    {
        // set the tid bit to indicate the word processed by tid is present
        atomicOr(&d_query_results[tid / 32], (1 << (tid % 32)));
    }
}

__global__ void map_bloom_kernel(char *d_words_to_insert, int len_words_to_insert, uint32_t *d_word_indices, uint32_t *d_bloom_filter, uint32_t total_words, int HASH_FUNCTION)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid >= total_words)
    {
        return;
    }

    // Find word start and end indices
    uint32_t word_len = d_word_indices[tid + 1] - d_word_indices[tid] - 1;
    uint32_t si = d_word_indices[tid] + 1;
    uint32_t ei = d_word_indices[tid + 1] - 1;

    // char *word = (char *)malloc((ei - si + 1) * sizeof(char));
    char word[128];

    for (uint32_t i = si; si <= ei; si++)
    {
        word[si - i] = d_words_to_insert[si];
    }

    // Add word to bloom filter
    bloom_insert(d_bloom_filter, word, word_len, tid, HASH_FUNCTION);
    // free(word);
}

__global__ void query_bloom_kernel(char *d_words_to_query, int len_words_to_insert, uint32_t *d_word_indices, uint32_t *d_bloom_filter, uint32_t *d_query_results, uint32_t total_words, int HASH_FUNCTION)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // printf("%d: query kernel\n", tid);

    if (tid >= total_words)
    {
        return;
    }

    // Find word start and end indices
    uint32_t word_len = d_word_indices[tid + 1] - d_word_indices[tid] - 1;
    uint32_t si = d_word_indices[tid] + 1;
    uint32_t ei = d_word_indices[tid + 1] - 1;

    // char *word = (char *)malloc((ei - si + 1) * sizeof(char));
    char word[128];

    for (uint32_t i = si; si <= ei; si++)
    {
        // printf("tid, bid, si: %d, %d, %d\n", threadIdx.x, blockIdx.x, si);
        word[si - i] = d_words_to_query[si];
    }

    // Add word to bloom filter
    bloom_query(d_bloom_filter, word, word_len, d_query_results, tid, HASH_FUNCTION);
    // free(word);
}

int main(int argc, char const *argv[])
{
    hipFree(0);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device name: %s\n", prop.name);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    HASH_FUNCTION = atoi(argv[1]);

    /* === Intialize profiling related variables === */
    // hipEvent_t start, stop;
    // float time;

    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    /* === Map Words to Bloom Filter === */

    char* words_to_insert = read_from_file(FILE_WORDS_TO_INSERT);
    uint32_t len_words_to_insert = strlen(words_to_insert);

    uint32_t *word_indices;
    int wi = -1;
    total_words = 0;

    uint32_t *bloom_filter = (uint32_t *)calloc(ceil(BLOOM_FILTER_SIZE / 32.0), sizeof(uint32_t));

    for (uint32_t i = 0; i < len_words_to_insert; i++)
    {
        if (words_to_insert[i] == ' ')
        {
            total_words++;
        }
    }
    // To account for the space at the beginning and the end
    total_words -= 1;

    // printf("\n---\n");
    // printf("Number of items to insert: %d\n", total_words);
    // printf("Number of bits in the Bloom Filter: %d\n", BLOOM_FILTER_SIZE);
    // printf("Number of hash functions: %d\n", N_HASHES);
    // printf("---\n");

    word_indices = (uint32_t *)calloc((total_words + 1), sizeof(uint32_t));

    for (uint32_t i = 0; i < len_words_to_insert; i++)
    {
        if (words_to_insert[i] == ' ')
        {
            word_indices[++wi] = i;
        }
    }

    // printf("\n == Before Inserting Words ==\n");
    // print_bloom_filter(bloom_filter);

    char *d_words_to_insert;
    uint32_t *d_word_indices;
    uint32_t *d_bloom_filter;

    hipEventRecord(start);
    hipMalloc((void **)&d_words_to_insert, len_words_to_insert * sizeof(char));
    hipMalloc((void **)&d_word_indices, (total_words + 1) * sizeof(uint32_t));
    hipMalloc((void **)&d_bloom_filter, ceil(BLOOM_FILTER_SIZE / 32.0) * sizeof(uint32_t));

    hipMemcpy(d_words_to_insert, words_to_insert, len_words_to_insert * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, (total_words + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_bloom_filter, bloom_filter, ceil(BLOOM_FILTER_SIZE / 32.0) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Transferring strings and bloom filter from Host to GPU Global Memory took %f ms\n", milliseconds);

    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("1.1.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    // hipEventRecord( start, 0 );

    hipEventRecord(start);
    map_bloom_kernel<<<ceil(total_words / 256.0), 256>>>(d_words_to_insert, len_words_to_insert, d_word_indices, d_bloom_filter, total_words, HASH_FUNCTION);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Inserting words to Bloom Filter (kernel) took:  %f ms\n", milliseconds);

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("1.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(start);
    hipMemcpy(bloom_filter, d_bloom_filter, ceil(BLOOM_FILTER_SIZE / 32.0) * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Transferring Bloom Filter from GPU Global Memory to Host took %f ms\n", milliseconds);


    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("2.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    // printf("\n == After Inserting Words to Bloom Filter ==\n");
    // print_bloom_filter(bloom_filter);

    // printf("== Inserting words using gpu took %f ms ==\n", time);

    hipFree(d_words_to_insert);
    hipFree(d_word_indices);

    /* === Query Bloom Filter === */

    char *words_to_query = read_from_file(FILE_WORDS_TO_QUERY);
    uint32_t len_words_to_query = strlen(words_to_query);

    total_words = 0;
    for (uint32_t i = 0; i < len_words_to_query; i++)
    {
        if (words_to_query[i] == ' ')
        {
            total_words++;
        }
    }
    // To account for the space at the beginning and the end
    total_words -= 1;

    word_indices = (uint32_t *)calloc((total_words + 1), sizeof(int));

    wi = -1;
    for (uint32_t i = 0; i < len_words_to_query; i++)
    {
        if (words_to_query[i] == ' ')
        {
            word_indices[++wi] = i;
        }
    }

    uint32_t *query_results = (uint32_t *)calloc(total_words, sizeof(uint32_t));
    uint32_t *d_query_results;
    char *d_words_to_query;

    hipEventRecord(start);

    hipMalloc((void **)&d_words_to_query, len_words_to_query * sizeof(char));
    hipMalloc((void **)&d_word_indices, (total_words + 1) * sizeof(int));
    hipMalloc((void **)&d_query_results, (ceil(total_words / 32.0) * sizeof(uint32_t)));


    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("4.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipMemcpy(d_words_to_query, words_to_query, len_words_to_query * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word_indices, word_indices, (total_words + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_query_results, query_results, (ceil(total_words / 32.0) * sizeof(uint32_t)), hipMemcpyHostToDevice);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Transerring Query Words to Global Memory took:  %f ms\n", milliseconds);

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("5.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(start);

    query_bloom_kernel<<<ceil(total_words / 256.0), 256>>>(d_words_to_query, len_words_to_query, d_word_indices, d_bloom_filter, d_query_results, total_words, HASH_FUNCTION);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Querying words from Bloom Filter (kernel) took:  %f ms\n", milliseconds);

    err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("6.CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(start);
    hipMemcpy(query_results, d_query_results, (ceil(total_words / 32.0) * sizeof(uint32_t)), hipMemcpyDeviceToHost);
        hipEventRecord(stop);

    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Transferring query results to host took:  %f ms\n", milliseconds);

    // printf("\n== After Querying Words from Bloom Filter ==\n");

    print_query_results(words_to_query, len_words_to_query, word_indices, query_results, total_words);

    hipFree(d_words_to_query);
    hipFree(d_word_indices);
    hipFree(d_query_results);
    hipFree(d_bloom_filter);

    return 0;
}